#include <stdio.h>
#include "matx.h"

void test_argmax(void)
{
  using cub_index_t = int;

  int num_segments = 4;
  auto t_in = matx::make_tensor<int>({num_segments, 2, 5});
  auto t_value_out = matx::make_tensor<int>({num_segments});
  auto t_index_out = matx::make_tensor<int>({num_segments});

  t_in.SetVals(
  {
    {{1, 2, 3, 4, 5},  {6, 7, 8, 9, 10}},
    {{1, 3, 5, 7, 10}, {2, 4, 6, 8, 9}},
    {{2, 4, 5, 1, 1},  {1, 1, 1, 1, 1}},
    {{INT_MIN, INT_MIN, INT_MIN, INT_MIN, INT_MIN},  {INT_MIN, INT_MIN, INT_MIN, INT_MIN, INT_MIN}},
  });
  cub_index_t golden_index[] {9, 4, 2, 0};
  int golden_value[] {10, 10, 5, INT_MIN};


  (matx::mtie(t_value_out, t_index_out) = argmax(t_in, {1, 2})).run();
  hipDeviceSynchronize();
  printf("MatX\n");
  for (int k=0; k<num_segments; k++)
  {
    printf("  [%d] %d => %d",k, t_index_out(k), t_value_out(k));
    if (golden_index[k] != t_index_out(k))
    {
      printf(" Index Mismatch (%d)",golden_index[k]);
    }
    if (golden_value[k] != t_value_out(k))
    {
      printf(" Value Mismatch (%d)",golden_value[k]);
    }
    printf("\n");
  }

  //matx::tensor_t<int, 0> t_max_value_out{{}};
  //matx::tensor_t<int, 0> t_max_index_out{{}};
  //(matx::mtie(t_max_value_out, t_max_index_out) = argmax(t_in)).run();
  //hipDeviceSynchronize();
  //matx::print(t_max_value_out);
  //matx::print(t_max_index_out);
}

int main(void)
{
  MATX_ENTER_HANDLER();

  test_argmax();

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
  return 0;
}
